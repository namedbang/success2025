#include "hip/hip_runtime.h"
/*
 * @Author: bangbang 1789228622@qq.com
 * @Date: 2024-11-10 16:21:41
 * @LastEditors: bangbang 1789228622@qq.com
 * @LastEditTime: 2025-01-09 22:35:29
 * @FilePath: /success2025/src/cuda/inRange_gpu.cu
 * @Description:
 *
 * Copyright (c) 2024 by CDTU-Success, All Rights Reserved.
 */

//---------------------inRange_gpu.cu-----start-----------
#include "./inRange_gpu.cuh"
#include <opencv2/core/cuda.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
struct RectF
{
    float x;
    float y;
    float width;
    float height;
};
__global__ void maskImageOutsideRect(cv::cuda::PtrStepSzb d_image, int width, int height, RectF *rects, int num_rects)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // 检查是否在图像范围内
    if (x < width && y < height)
    {
        bool inside_any_rect = false;

        // 遍历每个矩形框，检查当前像素是否在任意一个矩形框内
        for (int i = 0; i < num_rects; i++)
        {
            // 获取矩形框的 x, y, width, height
            int rect_x_min = static_cast<int>(rects[i].x);
            int rect_y_min = static_cast<int>(rects[i].y);
            int rect_x_max = static_cast<int>(rects[i].x + rects[i].width);
            int rect_y_max = static_cast<int>(rects[i].y + rects[i].height);

            // 如果像素在当前矩形框内
            if (x >= rect_x_min && x < rect_x_max && y >= rect_y_min && y < rect_y_max)
            {
                inside_any_rect = true;
                break; // 找到一个矩形框包含该像素，跳出循环
            }
        }

        // 如果像素不在任何矩形框内，将其设置为黑色
        if (!inside_any_rect)
        {
            d_image(y, x) = 0; // 设置该像素为黑色 (对于 3 通道图像，uchar3 的每个通道都为 0)
        }
    }
}

__global__ void inRange_kernel(const cv::cuda::PtrStepSz<uchar3> src, cv::cuda::PtrStepSzb dst,
                               int lbc0, int ubc0, int lbc1, int ubc1, int lbc2, int ubc2)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= src.cols || y >= src.rows)
        return;

    uchar3 v = src(y, x);
    if (v.x >= lbc0 && v.x <= ubc0 && v.y >= lbc1 && v.y <= ubc1 && v.z >= lbc2 && v.z <= ubc2)
        dst(y, x) = 255;
    else
        dst(y, x) = 0;
}

// void inRange_gpu(cv::cuda::GpuMat &src, cv::Scalar &lowerb, cv::Scalar &upperb,
//                  cv::cuda::GpuMat &dst)
// {
//     const int m = 32;
//     int numRows = src.rows, numCols = src.cols; // 行，列
//     if (numRows == 0 || numCols == 0)
//         return;
//     // Attention! Cols Vs. Rows are reversed
//     const dim3 gridSize(ceil((float)numCols / m), ceil((float)numRows / m), 1);
//     const dim3 blockSize(m, m, 1);

//     inRange_kernel<<<gridSize, blockSize>>>(src, dst, lowerb[0], upperb[0], lowerb[1], upperb[1],
//                                             lowerb[2], upperb[2]);
// }

void inRange_gpu(cv::cuda::GpuMat &src, cv::Scalar &lowerb, cv::Scalar &upperb,
                 cv::cuda::GpuMat &dst, std::vector<Object> rect, cv::cuda::Stream &stream)
{
    const int m = 32;
    int numRows = src.rows, numCols = src.cols; // 行，列
    if (numRows == 0 || numCols == 0)
        return;
    // Attention! Cols Vs. Rows are reversed
    const dim3 gridSize(ceil((float)numCols / m), ceil((float)numRows / m), 1);
    const dim3 blockSize(m, m, 1);

    hipStream_t cuStream = reinterpret_cast<hipStream_t>(stream.cudaPtr());
    // 异步执行内核，传入CUDA流
    // inRange_kernel<<<gridSize, blockSize, 0, cuStream>>>(src, dst, lowerb[0], upperb[0], lowerb[1], upperb[1],
    //                                                      lowerb[2], upperb[2]);
    inRange_kernel<<<gridSize, blockSize>>>(src, dst, lowerb[0], upperb[0], lowerb[1], upperb[1],
                                            lowerb[2], upperb[2]);
    std::vector<cv::Rect_<float>> rect_armor;
    for (uint16_t i = 0; i < rect.size(); i++)
    {
        if (rect[i].label == 7 || rect[i].label == 8 || rect[i].label == 6) // TODO换
            rect_armor.push_back(rect[i].rect);
    }
    std::vector<RectF> rects;
    rects.reserve(rect_armor.size());
    for (const auto &rect : rect_armor)
    {
        rects.push_back({rect.x, rect.y, rect.width, rect.height});
    }
    RectF *d_rects;
    size_t size = rects.size() * sizeof(RectF);
    hipMalloc(&d_rects, size);
    hipMemcpy(d_rects, rects.data(), size, hipMemcpyHostToDevice);
    int temp = rect_armor.size();
    maskImageOutsideRect<<<gridSize, blockSize>>>(dst, numCols, numRows, d_rects, temp);
}

//---------------------inRange_gpu.cu-----end-----------
